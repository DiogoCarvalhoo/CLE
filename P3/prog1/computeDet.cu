#include "hip/hip_runtime.h"
/**
 *  \file computeDet.c (implementation file)
 *
 *  \brief Problem name: Compute Matrix Determinant.
 *
 *  Concurrency based on CUDA with the approach of calculation of the determinant by columns, and comparison to CPU version.
 *
 *  How to compile: make all
 *  How to run: ./computeDet -f mat128_32.bin
 *
 *  \author Diogo Filipe Amaral Carvalho - 92969 - June 2022
 *  \author Rafael Ferreira Baptista - 93367 - June 2022
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <libgen.h>
#include <unistd.h> 

#include "common.h"
#include <hip/hip_runtime.h>

/* allusion to internal functions */

/** \brief function to compute determinant of a matrix in cpu */
static void calculate_determinant_cpu_kernel (double * matrix_pointer, double * determinant,
                                              unsigned int order_of_matrix);

/** \brief cuda kernel function to compute determinant of a matrix in gpu */
__global__ static void calculate_determinant_cuda_kernel (double * __restrict__ mat, double * __restrict__ determinants,
                                                          unsigned int n_sectors, unsigned int sector_size);

/** \brief function to get the time elapsed */                                              
static double get_delta_time(void);

/** \brief function responsible to present the program usage */
static void printUsage(char *cmdName);

/**
 *   main program
 */

int main (int argc, char **argv)
{
  printf("%s Starting...\n", argv[0]);
  if (sizeof (unsigned int) != (size_t) 4)
     return 1;                                             // it fails with prejudice if an integer does not have 4 bytes

  /* process command line arguments */

  int opt;            /* selected option */
  char *fName;        /* file name */

  opterr = 0;
  do
  {
      switch ((opt = getopt(argc, argv, "f:h")))
      {
      case 'f': /* file name */
          if (optarg[0] == '-')
          {
              fprintf(stderr, "%s: file name is missing\n", basename(argv[0]));
              printUsage(basename(argv[0]));

              return EXIT_FAILURE;
          }
          fName = optarg;
          break;
      case 'h': /* help mode */
          printUsage(basename(argv[0]));

          return EXIT_SUCCESS;
      case '?': /* invalid option */
          fprintf(stderr, "%s: invalid option\n", basename(argv[0]));
          printUsage(basename(argv[0]));

          return EXIT_FAILURE;
      case -1:
          
          break;
      }
  } while (opt != -1);
  if (argc == 1) 
  {
    fprintf(stderr, "%s: invalid format\n", basename(argv[0]));
    return EXIT_FAILURE;
  }

  /* open text file */

  FILE * fpointer;
  fpointer = fopen(fName, "rb");

  if (fpointer == NULL) {
      fprintf(stderr, "It occoured an error while openning file. \n");
      exit(EXIT_FAILURE);
  }

  /* read file content */

  int number_of_matrix;
  int order_of_matrix;
  
  // Read Number of Matrix from File
  if(fread(&number_of_matrix, sizeof(int), 1, fpointer) != 1)
      strerror(1);
  printf("Number of matrices to be read = %i \n", number_of_matrix);   

  // Read Order of Matrix from File
  if(fread(&order_of_matrix, sizeof(int), 1, fpointer) != 1)
      strerror(1);
  printf("Matrices order = %i \n", order_of_matrix);

  /* set up the device */

  int dev = 0;

  hipDeviceProp_t deviceProp;
  CHECK (hipGetDeviceProperties (&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK (hipSetDevice (dev));

  /* create memory areas in host and device memory where the matrices data and determinants will be stored */

  int mat_size = order_of_matrix * order_of_matrix * sizeof(double);
  size_t mat_area_size = number_of_matrix * mat_size;
  double * host_mat, * host_determinants;
  double * device_mat, * device_determinants;

  if ((mat_area_size + number_of_matrix * sizeof(double)) > (size_t) 1.3e9)
     { fprintf (stderr,"The GeForce GTX 1660 Ti cannot handle more than 5GB of memory!\n");
       exit (1);
     }
  printf ("Total mat size: %d\n", (int) mat_area_size);
  
  host_mat = (double *) malloc (mat_area_size);
  host_determinants = (double *) malloc (number_of_matrix*sizeof(double));
  CHECK (hipMalloc ((void **) &device_mat, mat_area_size));
  CHECK (hipMalloc ((void **) &device_determinants, number_of_matrix*sizeof(double)));

  /* initialize the host data */

  (void) get_delta_time ();

  // Read all the matrices from file
  if(fread(host_mat, mat_area_size, 1, fpointer) != 1)
      strerror(1);
  
  // Initialize determinants as 1
  for (int i = 0; i<number_of_matrix; i++) {
    host_determinants[i] = 1;
  }

  printf ("The initialization of host data took %.3e seconds\n",get_delta_time ());

  /* copy the host data to the device memory */

  (void) get_delta_time ();
  CHECK (hipMemcpy (device_mat, host_mat, mat_area_size, hipMemcpyHostToDevice));
  CHECK (hipMemcpy (device_determinants, host_determinants, number_of_matrix * sizeof(double), hipMemcpyHostToDevice));
  printf ("The transfer of %d bytes from the host to the device took %.3e seconds\n",
          (int) mat_area_size , get_delta_time ());

  /* run the computational kernel */

  unsigned int gridDimX,gridDimY,gridDimZ,blockDimX,blockDimY,blockDimZ;
  int n_sectors, sector_size;

  n_sectors = number_of_matrix * order_of_matrix;
  sector_size = order_of_matrix;
  blockDimX = order_of_matrix;
  blockDimY = 1 << 0;                                             // optimize!
  blockDimZ = 1 << 0;                                             // do not change!
  gridDimX = number_of_matrix;
  gridDimY = 1 << 0;                                              // optimize!
  gridDimZ = 1 << 0;                                              // do not change!

  dim3 grid (gridDimX, gridDimY, gridDimZ);
  dim3 block (blockDimX, blockDimY, blockDimZ);

  if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != n_sectors)
     { printf ("Wrong configuration!\n");
       return 1;
     }
  (void) get_delta_time ();
  calculate_determinant_cuda_kernel <<<grid, block>>> (device_mat, device_determinants, n_sectors, sector_size);
  CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
  CHECK (hipGetLastError ());                                 // check for kernel errors
  printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
         gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

  /* copy kernel result back to host side */

  double *determinants;
  determinants = (double *) malloc (number_of_matrix*sizeof(double));
  CHECK (hipMemcpy (determinants, device_determinants, number_of_matrix*sizeof(double), hipMemcpyDeviceToHost));
  printf ("The transfer of %d bytes from the device to the host took %.3e seconds\n",
          (int) mat_area_size, get_delta_time ());

  /* free device global memory */

  CHECK (hipFree (device_mat));
  CHECK (hipFree (device_determinants));

  /* reset the device */

  CHECK (hipDeviceReset ());

  /* compute the determinants on the CPU */

  (void) get_delta_time ();
  double *cpu_determinants;
  cpu_determinants = (double *) malloc (number_of_matrix*sizeof(double));
  
  for (int i = 0; i < number_of_matrix; i++) {
    cpu_determinants[i] = 1;
    calculate_determinant_cpu_kernel (host_mat + (i*order_of_matrix*order_of_matrix), &cpu_determinants[i], order_of_matrix);
  }
  printf("The cpu kernel took %.3e seconds to run (single core)\n",get_delta_time ());

  /* show final results */

  for (int i = 0; i < number_of_matrix; i++) {
    printf("Processing matrix %d \n", i + 1);
    printf("GPU Determinant: %.3e \nCPU Determinant: %.3e \n\n", determinants[i], cpu_determinants[i]);
  }

  /* compare results */
  
  for(int i = 0; i < number_of_matrix; i++) {
    if (fabs(determinants[i] / cpu_determinants[i]) > 1.0001 || fabs(determinants[i] / cpu_determinants[i]) < 0.9999 )
      { 
      printf ("Mismatch in matrix %d. GPU calculated %.4e CPU calculated %.4e \n", i, determinants[i], cpu_determinants[i]);
      exit(1);
      }
  }
  printf ("All is well!\n");

  /* free host memory */

  free (host_mat);
  free (host_determinants);
  free (determinants);

  return 0;
}

/**
 * @brief function to compute determinant of a matrix in cpu
 * 
 * @param matrix_pointer 
 * @param determinant 
 * @param order_of_matrix 
 */
static void calculate_determinant_cpu_kernel (double * matrix_pointer, double * determinant,
                                              unsigned int order_of_matrix)
{
  // Define matrix coeficients structure (2D Array)
  double matrix_coeficients[order_of_matrix][order_of_matrix];

  // Read Matrix Coefficients from buffer and save it to 2D array
  for (int l = 0; l < order_of_matrix; l++) {
      int line_offset = l * order_of_matrix;
      for (int column = 0; column < order_of_matrix; column++) {
          double coefficient = *(matrix_pointer + column + line_offset);
          matrix_coeficients[l][column] = coefficient;
      }
  }

  for (int l = 0; l < order_of_matrix-1; l++) {
      if (matrix_coeficients[l][l] != 0.0) {
          
          for(int k=l+1; k<order_of_matrix; k++){
              double term=matrix_coeficients[k][l]/ matrix_coeficients[l][l];
              for(int j=0; j< order_of_matrix; j++){
                  matrix_coeficients[k][j]=matrix_coeficients[k][j]-term*matrix_coeficients[l][j];
              }
          }

      } else {
          // Find column != 0.0 to be changed
          int columnToChange = -1;
          for (int j=l+1; j<order_of_matrix; j++) {
              if (matrix_coeficients[l][j] != 0.0) {
                  columnToChange = j;
              }
          }

          if (columnToChange == -1) {
              *determinant = 0;
              break;
          } else {
              int temp;
              for (int i = 0; i < order_of_matrix; ++i) {
                  temp = matrix_coeficients[i][l - 1];
                  matrix_coeficients[i][l - 1] = matrix_coeficients[i][columnToChange - 1];
                  matrix_coeficients[i][columnToChange - 1] = temp;
              }
          }

      }

  }

  // Calculate Determinant from upper triangular matrix (Multiply Diagonal Values)
  for(int l = 0; l < order_of_matrix; l++){
      *determinant = *determinant*matrix_coeficients[l][l];
  }

}

/**
 * @brief cuda kernel function to compute determinant of a matrix in gpu
 * 
 * @param mat 
 * @param determinants 
 * @param n_sectors 
 * @param sector_size 
 * @return * cuda 
 */
__global__ static void calculate_determinant_cuda_kernel (double * __restrict__ mat, double * __restrict__ determinants,
                                                          unsigned int n_sectors, unsigned int sector_size)
{
  /* compute the thread number */

  int bkx = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;         // block identifier
  int idx = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;   // thread identifier

  if (idx >= n_sectors)
     return;                                             // safety precaution

  /* adjust pointers */

  mat += bkx * sector_size * sector_size;

  /* start the iteration cycle */
  
  for (int i = 0; i<sector_size; i++) {

    // If the diagonal coefficient is 0 we need to find a column to change their value
    if (mat[i*sector_size + i] == 0) {
      int columnToChange = -1;

      for (int j=i+1; j<sector_size; j++) {
        if (mat[i*sector_size + j] != 0.0) {
          columnToChange = j;
        }
      }

      if (columnToChange == -1) {
          determinants[bkx] = 0;
          break;
      } else {
          double temp = mat[i + idx * sector_size];
          mat[i + idx * sector_size] = mat[columnToChange + idx * sector_size];
          mat[columnToChange + idx * sector_size] = temp;
      }

      __syncthreads(); // Synchronizing all threads to ensure that all threads compute the same term value
    }

    // Get the terms and apply the multiplication and somation of each element in column
    for (int line = i+1; line<sector_size; line++) {
      
      double term = - mat[line*sector_size + i] / mat[i*sector_size + i];

      __syncthreads(); // Synchronizing all threads to ensure that all threads compute the same term value

      // Update the values of all the coefficients in the column
      mat[line*sector_size + idx] = mat[line*sector_size + idx] + term * mat[i*sector_size + idx];
    }
    
    // If it is the first column of this iteration
    if (idx == i) {
      double coef = mat[ (idx*sector_size) + idx ];
      determinants[bkx] = determinants[bkx] * coef;
    }
  }

}

/**
 * @brief Get the elapsed time
 * 
 * @return double 
 */
static double get_delta_time(void)
{
  static struct timespec t0,t1;

  t0 = t1;
  if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
  {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}

/**
 *  \brief print usage.
 */
static void printUsage(char *cmdName)
{
    fprintf(stderr, "\nSynopsis: %s OPTIONS [filename]\n"
                    "  OPTIONS:\n"
                    "  -h      --- print this help\n"
                    "  -f      --- filename\n",
            cmdName);
}