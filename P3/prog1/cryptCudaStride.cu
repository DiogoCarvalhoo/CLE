#include "hip/hip_runtime.h"
/**
 *   Tomás Oliveira e Silva, November 2017
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <libgen.h>
#include <unistd.h> 

#include "common.h"
#include <hip/hip_runtime.h>

/* allusion to internal functions */

static void calculate_determinant_cpu_kernel (double * matrix_pointer, double * determinant,
                                              unsigned int order_of_matrix);
__global__ static void calculate_determinant_cuda_kernel (double * __restrict__ mat, double * __restrict__ determinants,
                                                          unsigned int n_sectors, unsigned int sector_size);
static double get_delta_time(void);

/** \brief function responsible to present the program usage */
static void printUsage(char *cmdName);

/**
 *   main program
 */

int main (int argc, char **argv)
{
  printf("%s Starting2...\n", argv[0]);
  if (sizeof (unsigned int) != (size_t) 4)
     return 1;                                             // it fails with prejudice if an integer does not have 4 bytes


  /* process command line arguments */

  int opt;            /* selected option */
  char *fName;   /* file name (initialized to "no name" by default) */

  opterr = 0;
  do
  {
      switch ((opt = getopt(argc, argv, "f:h")))
      {
      case 'f': /* file name */
          if (optarg[0] == '-')
          {
              fprintf(stderr, "%s: file name is missing\n", basename(argv[0]));
              printUsage(basename(argv[0]));

              return EXIT_FAILURE;
          }
          fName = optarg;
          break;
      case 'h': /* help mode */
          printUsage(basename(argv[0]));

          return EXIT_SUCCESS;
      case '?': /* invalid option */
          fprintf(stderr, "%s: invalid option\n", basename(argv[0]));
          printUsage(basename(argv[0]));

          return EXIT_FAILURE;
      case -1:
          
          break;
      }
  } while (opt != -1);
  if (argc == 1) 
  {
    fprintf(stderr, "%s: invalid format\n", basename(argv[0]));
    return EXIT_FAILURE;
  }

  /* open text file */

  FILE * fpointer;
  fpointer = fopen(fName, "rb");

  if (fpointer == NULL) {
      fprintf(stderr, "It occoured an error while openning file. \n");
      exit(EXIT_FAILURE);
  }

  /* read file content */

  int number_of_matrix;
  int order_of_matrix;
  
  // Read Number of Matrix from File
  if(fread(&number_of_matrix, sizeof(int), 1, fpointer) != 1)
      strerror(1);
  printf("Number of matrices to be read = %i \n", number_of_matrix);   

  // Read Order of Matrix from File
  if(fread(&order_of_matrix, sizeof(int), 1, fpointer) != 1)
      strerror(1);
  printf("Matrices order = %i \n", order_of_matrix);

  /* set up the device */

  int dev = 0;

  hipDeviceProp_t deviceProp;
  CHECK (hipGetDeviceProperties (&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK (hipSetDevice (dev));

  /* create memory areas in host and device memory where the disk sectors data and sector numbers will be stored */

  int mat_size = order_of_matrix * order_of_matrix * sizeof(double);
  size_t mat_area_size = number_of_matrix * mat_size;
  double * host_mat, * host_determinants;
  double * device_mat, * device_determinants;

  if ((mat_area_size + number_of_matrix * sizeof(double)) > (size_t) 1.3e9)
     { fprintf (stderr,"The GeForce GTX 1660 Ti cannot handle more than 5GB of memory!\n");
       exit (1);
     }
  printf ("Total mat size: %d\n", (int) mat_area_size);
  
  host_mat = (double *) malloc (mat_area_size);
  host_determinants = (double *) malloc (number_of_matrix*sizeof(double));
  CHECK (hipMalloc ((void **) &device_mat, mat_area_size));
  CHECK (hipMalloc ((void **) &device_determinants, number_of_matrix*sizeof(double)));

  /* initialize the host data */

  (void) get_delta_time ();

  if(fread(host_mat, mat_area_size, 1, fpointer) != 1)
      strerror(1);
  
  for (int i = 0; i<number_of_matrix; i++) {
    host_determinants[i] = 1;
  }

  printf ("The initialization of host data took %.3e seconds\n",get_delta_time ());

  /* copy the host data to the device memory */

  (void) get_delta_time ();
  CHECK (hipMemcpy (device_mat, host_mat, mat_area_size, hipMemcpyHostToDevice));
  CHECK (hipMemcpy (device_determinants, host_determinants, number_of_matrix * sizeof(double), hipMemcpyHostToDevice));
  printf ("The transfer of %d bytes from the host to the device took %.3e seconds\n",
          (int) mat_area_size , get_delta_time ());

  /* run the computational kernel */

  unsigned int gridDimX,gridDimY,gridDimZ,blockDimX,blockDimY,blockDimZ;
  int n_sectors, sector_size;

  n_sectors = number_of_matrix * order_of_matrix;
  sector_size = order_of_matrix;
  blockDimX = order_of_matrix;
  blockDimY = 1 << 0;                                             // optimize!
  blockDimZ = 1 << 0;                                             // do not change!
  gridDimX = number_of_matrix;
  gridDimY = 1 << 0;                                              // optimize!
  gridDimZ = 1 << 0;                                              // do not change!

  dim3 grid (gridDimX, gridDimY, gridDimZ);
  dim3 block (blockDimX, blockDimY, blockDimZ);

  if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != n_sectors)
     { printf ("Wrong configuration!\n");
       return 1;
     }
  (void) get_delta_time ();
  calculate_determinant_cuda_kernel <<<grid, block, sector_size-1>>> (device_mat, device_determinants, n_sectors, sector_size);
  CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
  CHECK (hipGetLastError ());                                 // check for kernel errors
  printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
         gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

  /* copy kernel result back to host side */

  double *modified_mat;
  double *determinants;

  //modified_device_sector_data = (unsigned int *) malloc (sector_data_size);
  modified_mat = (double *) malloc (mat_area_size);
  determinants = (double *) malloc (number_of_matrix*sizeof(double));
  CHECK (hipMemcpy (modified_mat, device_mat, mat_area_size, hipMemcpyDeviceToHost));
  CHECK (hipMemcpy (determinants, device_determinants, number_of_matrix*sizeof(double), hipMemcpyDeviceToHost));
  printf ("The transfer of %d bytes from the device to the host took %.3e seconds\n",
          (int) mat_area_size, get_delta_time ());

  for (int i = 0; i < number_of_matrix; i++) {
    printf("Processing matrix %d \n", i + 1);
    
    /*
    for (int k = 0; k < order_of_matrix; k++) {
      for (int l = 0; l < order_of_matrix; l++) {
        printf("%6.2f ", *(modified_mat + (k * order_of_matrix) + l + (i * order_of_matrix * order_of_matrix) ) );
      }
      printf("\n");
    }
    */
    
    printf("Determinant: %.3e \n\n", determinants[i]);
  }

  /* free device global memory */

  CHECK (hipFree (device_mat));
  //CHECK (hipFree (device_sector_number));

  /* reset the device */

  CHECK (hipDeviceReset ());

  /* compute the determinants on the CPU */

  (void) get_delta_time ();
  double *cpu_determinants;
  cpu_determinants = (double *) malloc (number_of_matrix*sizeof(double));
  
  for (int i = 0; i < number_of_matrix; i++) {
    cpu_determinants[i] = 1;
    calculate_determinant_cpu_kernel (host_mat + (i*order_of_matrix*order_of_matrix), &cpu_determinants[i], order_of_matrix);
  }
  printf("The cpu kernel took %.3e seconds to run (single core)\n",get_delta_time ());

  /* compare results */
  
  for(int i = 0; i < number_of_matrix; i++) {
    if (fabs(determinants[i] - cpu_determinants[i]) > 0.0001f)
       { 
        printf ("Mismatch in matrix %d. GPU calculated %.4e CPU calculated %.4e \n", i, determinants[i], cpu_determinants[i]);
        exit(1);
       }
  }
  printf ("All is well!\n");

  /* free host memory */

  free (host_mat);
  free (modified_mat);

  return 0;
}

static void calculate_determinant_cpu_kernel (double * matrix_pointer, double * determinant,
                                              unsigned int order_of_matrix)
{
  // Define matrix coeficients structure (2D Array)
  double matrix_coeficients[order_of_matrix][order_of_matrix];

  // Read Matrix Coefficients from buffer and save it to 2D array
  for (int l = 0; l < order_of_matrix; l++) {
      int line_offset = l * order_of_matrix;
      for (int column = 0; column < order_of_matrix; column++) {
          double coefficient = *(matrix_pointer + column + line_offset);
          matrix_coeficients[l][column] = coefficient;
      }
  }

  for (int l = 0; l < order_of_matrix-1; l++) {
      if (matrix_coeficients[l][l] != 0.0) {
          
          for(int k=l+1; k<order_of_matrix; k++){
              double  term=matrix_coeficients[k][l]/ matrix_coeficients[l][l];
              for(int j=0; j< order_of_matrix; j++){
                  matrix_coeficients[k][j]=matrix_coeficients[k][j]-term*matrix_coeficients[l][j];
              }
          }

      } else {
          // Find column != 0.0 to be changed
          int columnToChange = -1;
          for (int j=l+1; j<order_of_matrix; j++) {
              if (matrix_coeficients[l][j] != 0.0) {
                  columnToChange = j;
              }
          }

          if (columnToChange == -1) {
              *determinant = 0;
              break;
          } else {
              int temp;
              for (int i = 0; i < order_of_matrix; ++i) {
                  temp = matrix_coeficients[i][l - 1];
                  matrix_coeficients[i][l - 1] = matrix_coeficients[i][columnToChange - 1];
                  matrix_coeficients[i][columnToChange - 1] = temp;
              }
          }

      }

  }

  // Calculate Determinant from upper triangular matrix (Multiply Diagonal Values)
  for(int l = 0; l < order_of_matrix; l++){
      *determinant = *determinant*matrix_coeficients[l][l];
  }
}

__global__ static void calculate_determinant_cuda_kernel (double * __restrict__ mat, double * __restrict__ determinants,
                                                          unsigned int n_sectors, unsigned int sector_size)
{
  /* compute the thread number */

  int bkx = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;         // block identifier
  int idx = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;   // thread identifier

  if (idx >= n_sectors)
     return;                                             // safety precaution

  /* array to store the terms to update the coefficients in each iteration */
  extern __shared__ double term[];

  /* adjust pointers */

  mat += bkx * sector_size * sector_size;
  mat += idx;

  /* start the iteration cycle */

  for (int i = 0; i<sector_size-1; i++) {

    // If it is the first column of the iteration
    if (i == idx) {
      // Calculate the terms for each line and save them in the shared array
      for (int k = i+1; k < sector_size; k++) {
        term[k-1] = *(mat + (k*sector_size)) / *(mat + (i * sector_size));
      }
    }
    
    __syncthreads(); // Synchronizing all threads to get the current terms

    // Update the values of all the coefficients in the column
    for (int k = i+1; k<sector_size; k++) {
      *(mat + (k * sector_size)) = *(mat + (k * sector_size)) - term[k-1] * (*(mat + (i * sector_size)));
    }
    
  }

  // Thread 0 is responsible to calculate the determinant of this matrix
  if (idx == 0) {
    for(int l = 0; l < sector_size; l++){
        double coef = mat[ (l*sector_size) + l ];
        determinants[bkx] = determinants[bkx] * coef;
    }
  }

}

static double get_delta_time(void)
{
  static struct timespec t0,t1;

  t0 = t1;
  if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
  {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}

/**
 *  \brief print usage.
 */
static void printUsage(char *cmdName)
{
    fprintf(stderr, "\nSynopsis: %s OPTIONS [filename]\n"
                    "  OPTIONS:\n"
                    "  -h      --- print this help\n"
                    "  -f      --- filename\n",
            cmdName);
}